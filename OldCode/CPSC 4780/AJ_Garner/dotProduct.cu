/*
 * Author: AJ Garner
 * Class: CPSC 4780
 * Date: 09/15/2024
 *
 * Description:
 * This program computes the dot product of two large vectors both on the CPU and GPU. It compares the performance
 * of CPU and GPU implementations, including memory allocation, data transfer, and kernel execution on the GPU.
 * The results are printed along with the time taken for each step to analyze the speedup achieved using GPU parallel processing.
 *
 * Compilation Instructions:
 * First Make sure to run the module:
 *   module load cuda gcc
 *
 * To compile this program, use the NVIDIA CUDA compiler (nvcc):
 *   nvcc dotProduct.cu -o dotProduct
 *
 * Run the program:
 * After compiling, you can run the program using:
 *   ./dotProduct
 *
 * Note: If there is an error when trying to compile, such as the compiler may be unsupported, please compile using
 * this command instead: nvcc -allow-unsupported-compiler dotProduct.cu -o dotProduct
 */

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// Timer functions to measure different sections of the code
long long start_timer() {
    struct timeval current_time;
    gettimeofday(&current_time, NULL);
    return current_time.tv_sec * 1000000 + current_time.tv_usec;
}

// Stops the timer and prints the elapsed time for a specific label
long long stop_timer(long long start_time, char *label) {
    struct timeval current_time;
    gettimeofday(&current_time, NULL);
    long long end_time = current_time.tv_sec * 1000000 + current_time.tv_usec;
    printf("%s: %.5f sec\n", label, ((float)(end_time - start_time)) / (1000 * 1000));
    return end_time - start_time;
}

// Computes the dot product on the CPU
float CPU_big_dot(float *vec_A, float *vec_B, int vector_size) {
    float dot_product_result = 0.0;
    for (int i = 0; i < vector_size; i++) {
        // Simple element-wise multiplication and accumulation
        dot_product_result += vec_A[i] * vec_B[i];
    }
    return dot_product_result;
}

// GPU kernel to compute dot product in parallel
__global__ void GPU_dot_product_kernel(float *vec_A, float *vec_B, float *partial_result, int vector_size) {
    __shared__ float shared_memory[512];  // Using shared memory to speed up reduction
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int cache_index = threadIdx.x;
    float temp_sum = 0.0;

    // Each thread computes part of the dot product
    while (thread_id < vector_size) {
        temp_sum += vec_A[thread_id] * vec_B[thread_id];
        thread_id += blockDim.x * gridDim.x;
    }

    shared_memory[cache_index] = temp_sum;
    __syncthreads();

    // Perform reduction within the block
    int offset = blockDim.x / 2;
    while (offset != 0) {
        if (cache_index < offset) {
            shared_memory[cache_index] += shared_memory[cache_index + offset];
        }
        __syncthreads();
        offset /= 2;
    }

    // Use atomicAdd to accumulate the results across all blocks
    if (cache_index == 0) {
        atomicAdd(partial_result, shared_memory[0]);
    }
}

// Function to handle the GPU computation, including memory allocation and transfers
float GPU_big_dot(float *vec_A, float *vec_B, int vector_size) {
    float *device_vec_A, *device_vec_B, *device_partial_result;
    float host_result = 0.0;
    hipError_t cuda_status;

    // Allocate memory on the GPU
    long long timer_start = start_timer();
    cuda_status = hipMalloc((void **)&device_vec_A, vector_size * sizeof(float));
    if (cuda_status != hipSuccess) {
        printf("CUDA error in hipMalloc (device_vec_A): %s\n", hipGetErrorString(cuda_status));
        return -1;
    }
    cuda_status = hipMalloc((void **)&device_vec_B, vector_size * sizeof(float));
    if (cuda_status != hipSuccess) {
        printf("CUDA error in hipMalloc (device_vec_B): %s\n", hipGetErrorString(cuda_status));
        return -1;
    }
    cuda_status = hipMalloc((void **)&device_partial_result, sizeof(float));
    if (cuda_status != hipSuccess) {
        printf("CUDA error in hipMalloc (device_partial_result): %s\n", hipGetErrorString(cuda_status));
        return -1;
    }
    stop_timer(timer_start, "Memory allocation on GPU");

    // Transfer data from host to device (CPU to GPU)
    timer_start = start_timer();
    hipMemcpy(device_vec_A, vec_A, vector_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_vec_B, vec_B, vector_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_partial_result, &host_result, sizeof(float), hipMemcpyHostToDevice);
    stop_timer(timer_start, "Memory transfer to GPU");

    // Launch the kernel with an appropriate number of blocks and threads
    int threads_per_block = 512;
    int num_blocks = (vector_size + threads_per_block - 1) / threads_per_block;
    timer_start = start_timer();
    GPU_dot_product_kernel<<<num_blocks, threads_per_block>>>(device_vec_A, device_vec_B, device_partial_result, vector_size);
    
    // Ensure the kernel has completed
    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        printf("CUDA error in kernel execution: %s\n", hipGetErrorString(cuda_status));
        return -1;
    }
    stop_timer(timer_start, "Kernel execution");

    // Copy the result from device to host (GPU to CPU)
    timer_start = start_timer();
    hipMemcpy(&host_result, device_partial_result, sizeof(float), hipMemcpyDeviceToHost);
    stop_timer(timer_start, "Memory transfer to CPU");

    // Free up GPU memory
    hipFree(device_vec_A);
    hipFree(device_vec_B);
    hipFree(device_partial_result);

    return host_result;
}

int main() {
    int vector_size = 1024 * 1024;
    float *host_vec_A = (float *)malloc(vector_size * sizeof(float));
    float *host_vec_B = (float *)malloc(vector_size * sizeof(float));

    // Initialize vectors with random values
    for (int i = 0; i < vector_size; i++) {
        host_vec_A[i] = (float)rand() / RAND_MAX;
        host_vec_B[i] = (float)rand() / RAND_MAX;
    }

    // Perform dot product on the CPU
    long long timer_start = start_timer();
    float cpu_result = CPU_big_dot(host_vec_A, host_vec_B, vector_size);
    long long cpu_time = stop_timer(timer_start, "CPU computation");

    // Perform dot product on the GPU
    timer_start = start_timer();
    float gpu_result = GPU_big_dot(host_vec_A, host_vec_B, vector_size);
    long long gpu_time = stop_timer(timer_start, "Total GPU computation");

    // Output results and speedup
    printf("CPU result: %f\n", cpu_result);
    printf("GPU result: %f\n", gpu_result);
    printf("Difference: %f\n", abs(cpu_result - gpu_result));
    printf("Speedup (CPU time / GPU time): %.5f\n", (float)cpu_time / gpu_time);

    // Clean up memory
    free(host_vec_A);
    free(host_vec_B);

    return 0;
}